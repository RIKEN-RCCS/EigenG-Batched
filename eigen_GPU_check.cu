#include "hip/hip_runtime.h"
#include "gpu_arch.h"
#include "misc_gpu.hpp"

template <class T>
__device__ void
check_(const int id, const int nm, const int n, T *a_, const int m, const T *d_, const T *z_)
{
  const int myid = threadIdx.x % WARP_GPU_SIZE + 1;
  sync_over_warp();
#define	a(row,col)	(*(a_+(row-1)+(col-1)*nm))
#define	z(row,col)	(*(z_+(row-1)+(col-1)*nm))
#define	d(index)	(*(d_+(index-1)))

  const double ZERO = static_cast<double>(0);
  const double ONE  = static_cast<double>(1);

#if 0
  double pi = 4*atan(ONE);

  for (int i=myid; i<=n; i++) {
    const double theta = (2*(n+1-i)-1) * pi / (2*n+1);
    const double r = ONE / (2 * (ONE-cos(theta)));
    const double err = Abs((double)d(i)-r);
    const double EPS  = (ONE / (1<<26)) / (1<<26);
    const double eps  = (ONE / (1<<23));
    const double TOL = 256*(double)(std::is_same<T,double>::value?EPS:eps);
    if ( err > Abs((double)d(n))*TOL ) {
      printf("d:=%le ext:=%le res=%le\n",
        (double)d(i), r, err/Abs((double)d(n)));
    }
  }
#endif

  double err1 = ZERO;
  for (int i=1; i<=m; i++) {
    for (int k=myid; k<=n; k+=WARP_GPU_SIZE) {
      double ek = (double)ZERO;
      for (int j=1; j<=n; j++) {
        ek += (double)a(k,j) * (double)z(j,i);
      }
      const double dxk = (double)d(i)*(double)z(k,i);
      const double r = ek - dxk;
      err1 = err1 + r * r;
    }
  } sum_over_warp(err1);
  err1 = Sqrt(err1);

  double err2 = ZERO;
  for (int i=1; i<=m; i++) {
    for (int j=i; j<=m; j++) {
      double r = ZERO;
      for (int k=myid; k<=n; k+=WARP_GPU_SIZE) {
        r += (double)z(k,i)*(double)z(k,j);
      } sum_over_warp(r);
      const double t = (i==j ? ONE : ZERO);
      const double c = (i==j ? ONE: 2*ONE);
      r -= t;
      r *= r;
      err2 += c * r;
    }
  } sum_over_warp(err2);
  err2 = Sqrt(err2);

  if ( myid == 1 ) {
    const double D_left = fabs((double)d(1));
    const double D_right = fabs((double)d(n));
    const double Dmax = fmax(D_left, D_right);
    const double EPS  = (ONE / (1<<26)) / (1<<26);
    const double eps  = (ONE / (1<<23));
//    const double TOL = (std::is_same<T,double>::value?EPS*512:eps*16)*sqrt((double)(n+24));
    const double TOL = (std::is_same<T,double>::value?EPS*(2*512-1):eps*(2*16-1))*sqrt((double)(n+16));

    double e1 = (( err1 > Dmax*TOL ) ? (err1/Dmax) : ZERO);
    double e2 = (( err2 > n*TOL ) ? err2 : ZERO);

    float * ans = (float *)(a_);
    ans[0] = (float)(e1);
    ans[1] = (float)(e2);
    ans[2] = (float)(TOL);
    ans[3] = (float)(std::is_same<T,double>::value?EPS:eps);
  }

#undef	a
#undef	z
#undef	d
#undef	e
  sync_over_warp();
}

template <class T>
__global__ void
parallel_check_(const int L, const int nm, const int n, T *a_, const int m, const T *d_, const T *z_)
{
  const int pos = (threadIdx.x+blockIdx.x*blockDim.x)/WARP_GPU_SIZE;
  const int step = (blockDim.x*gridDim.x)/WARP_GPU_SIZE;
  for(int id=pos; id<L; id+=step){
    T *a = (T *)a_ + (size_t)id*nm*n;
    T *d = (T *)d_ + (size_t)id*n;
    T *z = (T *)z_ + (size_t)id*nm*n;
    check_(L, nm, n, a, m, d, z);
  }
}

#if defined(__NVCC__)
template <class T>
__global__ void
print_logs_(const int L, const int nm, const int n, const T *a_, const int m, const T *d_, const T *z_)
{
  const double ZERO = static_cast<double>(0);
//  const double ONE  = static_cast<double>(1);

#define	a(row,col)	(*(a_+(row-1)+(col-1)*nm))
#define	z(row,col)	(*(z_+(row-1)+(col-1)*nm))
#define	d(index)	(*(d_+(index-1)))

  for(int id=0; id<L; id++){
    T *a = (T *)a_ + (size_t)id*nm*n;
    T *z = (T *)z_ + (size_t)id*nm*n;
    T *d = (T *)d_ + (size_t)id*n;

    float * ans = (float *)(a);
    double err1 = (double)(ans[0]);
    double err2 = (double)(ans[1]);
    double TOL  = (double)(ans[2]);
    double eps  = (double)(ans[3]);

    if ( err1 != ZERO ) {
      printf("[%06d] Accuracy error in the relative resisdual (||Ax-dx||_F=%le/%le/%le)\n", id, err1,TOL,eps);
    }
    if ( err2 != ZERO ) {
      printf("[%06d] Orthonormality error (||ZZ-I||_F=%le/%le/%le)\n", id, err2,TOL,eps);
    }

  }

//  printf("Eigen %le / %le\n", (double)d_[0],(double)d_[n-1]);

#undef	a
#undef	z
#undef	d
}
#endif

template <class T>
__host__ gpuError_t
eigen_GPU_check_RUN(const int L, const int nm, const int n, const int m, T *a_, T *w_, T *z_, const gpuStream_t stream)
{
  parallel_check_ <T> <<< L, WARP_GPU_SIZE, 0, stream >>> (L, nm, n, a_, m, w_, z_);
#if defined(__HIPCC__)
{
  const double ZERO = static_cast<double>(0);
  const double ONE  = static_cast<double>(1);

  size_t len = sizeof(T)*L*nm*n;
  T *a_h = (T *)malloc(len);
  if ( a_h == NULL ) { return gpuErrorInvalidValue; }
//  gpuMemcpy(a_h, w_, sizeof(T)*n, gpuMemcpyDeviceToHost);
//  printf("Eigen %le / %le\n", (double)a_h[0],(double)a_h[n-1]);
  gpuMemcpy(a_h, a_, len, gpuMemcpyDeviceToHost);
  for(int id=0;id<L;id++) {
    float *ans = (float *)(a_h + (size_t)nm*n*id);
    double err1 = (double)(ans[0]);
    double err2 = (double)(ans[1]);
    double TOL  = (double)(ans[2]);
    double eps  = (double)(ans[3]);
    if ( err1 != ZERO ) {
      printf("[%06d] Accuracy error in the relative resisdual (||Ax-dx||_F=%le/%le/%le)\n", id, err1,TOL,eps);
    }
    if ( err2 != ZERO ) {
      printf("[%06d] Orthonormality error (||ZZ-I||_F=%le/%le/%le)\n", id, err2,TOL,eps);
    }
  }
  free(a_h);
}
#endif
#if defined(__NVCC__)
  print_logs_ <T> <<< 1, 1, 0, stream >>> (L, nm, n, a_, m, w_, z_);
#endif

  return gpuSuccess;
}

extern "C" {

__host__ void
eigen_GPU_check_DP(const int L, const int nm, const int n, const int m, double *a_, double *w_, double *z_, const gpuStream_t stream)
{
  eigen_GPU_check_RUN <double>(L, nm, n, m, a_, w_, z_, stream);
}

__host__ void
eigen_GPU_check_FP(const int L, const int nm, const int n, const int m, float *a_, float *w_, float *z_, const gpuStream_t stream)
{
  eigen_GPU_check_RUN <float>(L, nm, n, m, a_, w_, z_, stream);
}

}

